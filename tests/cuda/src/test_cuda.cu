#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include <unistd.h>
#include <stdint.h>
#include <assert.h>

#include <mma.h>
#include <iostream>

#define CUDACHECK(cmd)                                         \
    do                                                         \
    {                                                          \
        hipError_t e = cmd;                                   \
        if (e != hipSuccess)                                  \
        {                                                      \
            printf("Failed: Cuda error %s:%d '%s'\n",          \
                   __FILE__, __LINE__, hipGetErrorString(e)); \
            exit(EXIT_FAILURE);                                \
        }                                                      \
    } while (0)

#define NCCLCHECK(cmd)                                         \
    do                                                         \
    {                                                          \
        ncclResult_t r = cmd;                                  \
        if (r != ncclSuccess)                                  \
        {                                                      \
            printf("Failed, NCCL error %s:%d '%s'\n",          \
                   __FILE__, __LINE__, ncclGetErrorString(r)); \
            exit(EXIT_FAILURE);                                \
        }                                                      \
    } while (0)

int main(int argc, char *argv[])
{
    int dev = 0;
    // cudaDeviceProp deviceProp;
    // cudaGetDeviceProperties(&deviceProp, dev);

    // printf("%lu\n", deviceProp.sharedMemPerMultiprocessor);
    printf("%lu\n", sizeof(uint8_t));

    return 0;
}
