#include "hip/hip_runtime.h"
#include <iostream>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
using namespace std;

__global__ void device_kernel(at::Half *hh)
{
    half y = __float2half(hh[0]);
    half t = __hmul(y, y);
    hh[0] = __half2float(t);
}

void zeyu_cuda_tests()
{
    at::Half x = 3.1786;

    at::Half y = x * x;
    cout << y << "\n";

    at::Half *d_o;
    at::Half *h_o = new at::Half[1];
    hipMalloc((void **)&d_o, 1 * sizeof(at::Half));
    h_o[0] = 3.1786;
    hipMemcpy(d_o, h_o, 1 * sizeof(at::Half), hipMemcpyHostToDevice);
    device_kernel<<<1, 1>>>(d_o);

    hipMemcpy(h_o, d_o, 1 * sizeof(at::Half), hipMemcpyDeviceToHost);
    cout << h_o[0] << "\n";
    hipFree(d_o);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    m.def("call", &zeyu_cuda_tests, "Zeyu CUDA Tests");
}
